#include<stdio.h>
#include<hip/hip_runtime.h>
__global__ void display(int n){
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if(idx<n){
        printf("Hello World in cuda Programming!!! from thread : %d\n",idx);
    }
}
int main(){
    int n;
    int threadsPerBlock,blocksPerGrid;
    printf("Enter number of messages : ");
    scanf("%d",&n);
    printf("Enter the number of threads per block: ");
    scanf("%d", &threadsPerBlock);
    blocksPerGrid = ((n+threadsPerBlock-1)/threadsPerBlock);
    display<<<blocksPerGrid,threadsPerBlock>>>(n);
    hipDeviceSynchronize();
    return 0;
}

// to compile - nvcc add.cu -o add
// to run - ./add or add.exe