#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 512

__global__ void dotProduct(int *a, int *b, int *result, int n)
{
    __shared__ int partialSum[256];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int threadID = threadIdx.x;
    partialSum[threadID] = 0;

    if (tid < n)
    {
        partialSum[threadID] = a[tid] * b[tid];
    }

    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride /= 2)
    {
        if (threadID < stride)
        {
            partialSum[threadID] += partialSum[threadID + stride];
        }
        __syncthreads();
    }

    if (threadID == 0)
    {
        atomicAdd(result, partialSum[0]);
    }
}

int main()
{
    int host_a[N], host_b[N], host_result = 0;
    int *dev_a, *dev_b, *dev_result;

    for (int i = 0; i < N; i++)
    {
        host_a[i] = i + 1;
        host_b[i] = i + 2;
    }

    size_t size = N * sizeof(int);
    hipMalloc((void **)&dev_a, size);
    hipMalloc((void **)&dev_b, size);
    hipMalloc((void **)&dev_result, sizeof(int));

    hipMemcpy(dev_a, host_a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_result, &host_result, sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    dotProduct<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_result, N);

    hipMemcpy(&host_result, dev_result, sizeof(int), hipMemcpyDeviceToHost);

    printf("Dot product: %d\n", host_result);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_result);

    return 0;
}

// to compile - nvcc add.cu -o add
// to run - ./add or add.exe