#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 512

__global__ void matrixMultiply(int *A, int *B, int *C, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n)
    {
        int sum = 0;
        for (int k = 0; k < n; k++)
        {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

int main()
{
    int host_A[N][N], host_B[N][N], host_C[N][N];
    int *dev_A, *dev_B, *dev_C;

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            host_A[i][j] = i + j;
            host_B[i][j] = i - j;
        }
    }

    size_t size = N * N * sizeof(int);
    hipMalloc((void **)&dev_A, size);
    hipMalloc((void **)&dev_B, size);
    hipMalloc((void **)&dev_C, size);

    hipMemcpy(dev_A, host_A, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_B, host_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrixMultiply<<<blocksPerGrid, threadsPerBlock>>>(dev_A, dev_B, dev_C, N);

    hipMemcpy(host_C, dev_C, size, hipMemcpyDeviceToHost);

    printf("Result matrix (first 5x5 block):\n");
    for (int i = 0; i < 5; i++)
    {
        for (int j = 0; j < 5; j++)
        {
            printf("%d ", host_C[i][j]);
        }
        printf("\n");
    }

    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);

    return 0;
}

// to compile - nvcc add.cu -o add
// to run - ./add or add.exe