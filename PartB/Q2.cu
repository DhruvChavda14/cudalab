#include<stdio.h>
#include<hip/hip_runtime.h>
#define N 512
__global__ void addVectors(int *a,int *b, int *res, int n){
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if(idx<n){
        res[idx] = a[idx] + b[idx];
    }
}

int main(){
    int host_a[N],host_b[N], host_res[N];
    int *dev_a,*dev_b,*dev_res;
    for(int i=0;i<N;i++){
        host_a[i]=i;
        host_b[i]=N-i;
    }
    hipMalloc((void**)&dev_a,N*sizeof(int));
    hipMalloc((void**)&dev_b,N*sizeof(int));
    hipMalloc((void**)&dev_res,N*sizeof(int));
    hipMemcpy(dev_a, host_a, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, sizeof(int) * N, hipMemcpyHostToDevice);
    int threadsPerBlock = 256;
    int blockPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    addVectors<<<blockPerGrid, threadsPerBlock>>>(dev_a,dev_b,dev_res,N);
    hipMemcpy(host_res,dev_res,N*sizeof(int),hipMemcpyHostToDevice);
    printf("Result (Top 10 results) : \n");
    for(int i=0;i<10;i++){
        printf("c[%d] : %d \n",i,host_res[i]);
    }
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_res);
    return 0;
}

// to compile - nvcc add.cu -o add
// to run - ./add or add.exe