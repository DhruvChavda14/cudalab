#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
__global__ void addNum(int* a,int *b, int* res){
    *res = *a + *b;
}
int main(){
    int host_a=5,host_b=10,host_res;
    int *deva,*devb,*devres;
    hipMalloc((void**)&deva,sizeof(int));
    hipMalloc((void**)&devb,sizeof(int));
    hipMalloc((void**)&devres,sizeof(int));
    hipMemcpy(deva,host_a,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(devb,host_b,sizeof(int),hipMemcpyHostToDevice);
    addNum<<<1,1>>>(deva,devb,devres);
    hipMemcpyHostToDevice(&host_res, devres, sizeof(int), hipMemcpyHostToDevice);
    printf("%d + %d = %d\n",host_a,host_b,host_res);
    hipFree(deva);
    hipFree(devb);
    hipFree(devres);
}

// to compile - nvcc add.cu -o add
// to run - ./add or add.exe
